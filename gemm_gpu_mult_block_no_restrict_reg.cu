#include "hip/hip_runtime.h"
#include "gemm_gpu_mult_block.h"

#include <hip/hip_runtime_api.h>

// gemm_gpu_mult_block_no_restrict_reg - GEMM on GPU, using many blocks
// and without the __restrict__ keyword
// and stores the intermediate results in a register
// The block size is N
// The grid size is M
__global__
void gemm_gpu_mult_block_no_restrict_reg_kernel(
	int* C,		// [n, m], on gpu
	const int* A,	// [n, k], on gpu
	const int* B,	// [k, m], on gpu
	const int n,
	const int m,
	const int k
) {
	const int i = threadIdx.x;
	const int j = blockIdx.x;
	int res = 0;
	for (int l = 0; l < k; ++l) {
		res += A[i * k + l] * B[l * m + j];
	}
	C[i * m + j] = res;
}

void gemm_gpu_mult_block_no_restrict_reg(
	int* C,		// [n, m], on gpu
	const int* A,	// [n, k], on gpu
	const int* B,	// [k, m], on gpu
	const int n,
	const int m,
	const int k
) {
	gemm_gpu_mult_block_no_restrict_reg_kernel<<<m, n>>>(C, A, B, n, m, k);
}
